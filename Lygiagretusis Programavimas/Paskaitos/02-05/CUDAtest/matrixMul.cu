#include "hip/hip_runtime.h"
/*
* Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation and
* any modifications thereto.  Any use, reproduction, disclosure, or distribution
* of this software and related documentation without an express license
* agreement from NVIDIA Corporation is strictly prohibited.
*
*/

/* Matrix multiplication: C = A * B.
* Host code.
*
* This sample implements matrix multiplication and is exactly the same as
* Chapter 7 of the programming guide.
* It has been written for clarity of exposition to illustrate various CUDA
* programming principles, not with the goal of providing the most
* performant generic kernel for matrix multiplication.
*
* CUBLAS provides high-performance matrix multiplication.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <omp.h>

// includes, project
#include "matrixMul.h"
// includes, kernels
#include "matrixMul_kernel.cuh"
#include "matrixMul_naive.cuh"
#include "matrixMul_tiling.cuh"
#include "matrixMul_coalescing.cuh"
#include "matrixMul_noBankConflict.cuh"
#include "matrixMul_compOpt.cuh"
#include "matrixMul_unroll.cuh"
#include "matrixMul_prefetch.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int testNum);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

//extern "C"
//void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

void computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a = A[i * wA + k];
				double b = B[k * wB + j];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	int devID = 0;
	//hipSetDevice(devID);
	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	for(int i = 0;i<=7;i++)
		runTest(i);

	exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int testNum)
{

	/****************************************************/
	/*  Preparations                                    */
	/****************************************************/

	printf("[CUDA Matrix Multiply Using Version %d ] - Starting ...\n", testNum);

	// utilities
	hipEvent_t start;
	hipEvent_t stop;
	float msecTotal;


	// allocate host memory for matrices A and B
	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);//galima naudoti ir new float[size_A]
	unsigned int size_B = WB * HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);
	float flop = 2 * (float)WC * (float)HC * (float)WA;

	// set seed for rand()
	srand(2017);
	// initialize host memory
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);

	// allocate device memory
	float* d_A;
	hipMalloc((void**)&d_A, mem_size_A);
	float* d_B;
	hipMalloc((void**)&d_B, mem_size_B);

	// allocate device memory for result
	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* d_C;
	hipMalloc((void**)&d_C, mem_size_C);

	// allocate host memory for the result
	float* h_C = (float*)malloc(mem_size_C);

#if CHECK_RESULT == 1
	// create and start timer
	/*hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, NULL);*/
	double t1 = omp_get_wtime();
	// compute reference solution
	float* reference = (float*)malloc(mem_size_C);
	computeGold(reference, h_A, h_B, HA, WA, WB);
	// stop and destroy timer
	double t2 = omp_get_wtime();
	/*hipEventRecord(stop, NULL);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);*/
	msecTotal =1000.* (t2 - t1) ;
	//printf("Naive CPU (Golden Reference)\n");
	printf("CPU processing time: %f (ms), GFLOPS: %f \n", msecTotal, ((flop/1.e+9) / (msecTotal / 1.e+3)));
	//printf("Processing time2: %f (ms), \n", t2-t1);
#endif

	dim3 threads, grid;

	/****************************************************/
	/*  CUDA SDK example                                */
	/****************************************************/

	// create and start timer
	// copy host memory to device
	// setup execution parameters
	/*hipEventCreate(&start);
	hipEventRecord(start, NULL);*/
	t1 = omp_get_wtime();
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
	grid = dim3(WC / threads.x, HC / threads.y);

	// execute the kernel
	switch (testNum) {
	case 0:
		matrixMul <<< grid, threads >>> (d_C, d_A, d_B, WA, WB);
		break;
	case 1:
		matrixMul_naive <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 2:
		matrixMul_tiling <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 3:
		matrixMul_coalescing <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 4:
		matrixMul_noBankConflict <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 5:
		threads = dim3(BLOCK_SIZE, 4);
		grid = dim3(WC / (BLOCK_SIZE * 4), HC / BLOCK_SIZE);
		matrixMul_compOpt <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 6:
		threads = dim3(BLOCK_SIZE, 4);
		grid = dim3(WC / (BLOCK_SIZE * 4), HC / BLOCK_SIZE);
		matrixMul_unroll <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	case 7:
		threads = dim3(BLOCK_SIZE, 4);
		grid = dim3(WC / (BLOCK_SIZE * 4), HC / BLOCK_SIZE);
		matrixMul_prefetch <<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		break;
	}
	// copy result from device to host
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
	/*hipEventCreate(&stop);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);*/
	// stop and destroy timer
	//hipEventElapsedTime(&msecTotal, start, stop);
	t2 = omp_get_wtime();
	msecTotal =1000.*(t2-t1);
	//printf("GPU SDK Sample\n");
	printf("Processing time: %f (ms), GFLOPS: %f \n\n", msecTotal, flop / msecTotal / 1e+6);
#if CHECK_RESULT == 1
	// check result
	printDiff(reference, h_C, WC, HC);
#endif

	/****************************************************/
	/*  Cleaning                                        */
	/****************************************************/

	// clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
#if CHECK_RESULT == 1
	free(reference);
#endif
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	//hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
	for (int i = 0; i < size; ++i)
		data[i] = ((float)rand()) / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
	int i, j, k;
	int error_count = 0;
	for (j = 0; j<height; j++) {
		for (i = 0; i<width; i++) {
			k = j*width + i;
			if (fabs(data1[k] - data2[k]) > 0.1) {
				printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f \n", i, j, data1[k], data2[k]);
				error_count++;
				if(error_count>2)
					return;
			}
		}
	}
	printf("Total Errors = %d \n", error_count);
}
